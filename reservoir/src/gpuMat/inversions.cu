#include "hip/hip_runtime.h"

/**
 * \file inversions.cu
 * \brief defines cuda matrix inversion functions/
 * \author Florian Lance
 * \date 01/10/14
 */

#include "cula.h"

#include "gpuMat/configCuda.h"


#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/copy.h>


int culaWarmup(int gpuDevice)
{
    int cudaMinimumVersion   = culaGetCudaMinimumVersion();
    int cudaRuntimeVersion   = culaGetCudaRuntimeVersion();
    int cudaDriverVersion    = culaGetCudaDriverVersion();
    int cublasMinimumVersion = culaGetCublasMinimumVersion();
    int cublasRuntimeVersion = culaGetCublasRuntimeVersion();

    if(cudaRuntimeVersion < cudaMinimumVersion)
    {
        printf("-1\n %d", cudaMinimumVersion);
        return -1;
    }

    if(cudaDriverVersion < cudaMinimumVersion)
    {
        printf("-2\n %d", cudaMinimumVersion);
        return -1;
    }

    if(cublasRuntimeVersion < cublasMinimumVersion)
    {
        printf("-3\n %d", cublasMinimumVersion);
        return -1;
    }

    culaStatus l_oStatus;
    char l_buf[256];
    int l_info;

    culaSelectDevice(gpuDevice);
    l_oStatus = culaInitialize();

    if(l_oStatus != culaNoError)
    {
        l_info = culaGetErrorInfo();

        culaGetErrorInfoString(l_oStatus, l_info, l_buf, sizeof(l_buf));

        printf("%s\n", culaGetStatusString(l_oStatus));
        printf("%s\n", l_buf);
        return -1;
    }

    return 0;
}

void culaStop()
{
    culaShutdown();
}

// ############################################################################################# SVD DECOMPOSITION

bool svdDecomposition(float* data, int m, int n, float* S, float* VT, float* U)
{
    const int l_count = m * n;
    const int l_minDim = std::min(m,n);
    const char l_jobu  = 'A';
    const char l_jobvt = 'A';

    culaStatus l_status;

    thrust::device_vector<float> l_data(data, data + l_count);
    thrust::device_vector<float> l_U(m * m);
    thrust::device_vector<float> l_sigma(l_minDim);
    thrust::device_vector<float> l_Vt(n * n);

    if ((l_status = culaDeviceSgesvd(l_jobu, l_jobvt, m, n, l_data.data().get(), m, l_sigma.data().get(),l_U.data().get(), m, l_Vt.data().get(), n)) != culaNoError)
    {
        printf("%s\n", culaGetStatusString(l_status));
        delete[] data;
        data = NULL;
        return false;
    }

    delete[] data;
    data = NULL;

    thrust::copy(l_U.begin(), l_U.end(), U);
    thrust::copy(l_Vt.begin(), l_Vt.end(), VT);
    thrust::copy(l_sigma.begin(), l_sigma.end(), S);

    return true;
}

bool svdDecomposition_all(float* dataOverwrittenVt, int m, int n, float* S, float* U)
{
    const int l_count = m * n;
    const int l_minDim = std::min(m,n);

    culaStatus l_status;
    thrust::device_vector<float> l_dataOvt(dataOverwrittenVt, dataOverwrittenVt + l_count);
    thrust::device_vector<float> l_U(m * m);
    thrust::device_vector<float> l_sigma(l_minDim);

    if ((l_status = culaDeviceSgesvd('A', 'O', m, n, l_dataOvt.data().get(), m, l_sigma.data().get(),l_U.data().get(), m, NULL, n)) != culaNoError)
    {
        printf("%s\n", culaGetStatusString(l_status));
        return false;
    }

    thrust::copy(l_dataOvt.begin(), l_dataOvt.end(), dataOverwrittenVt);
    thrust::copy(l_U.begin(), l_U.end(), U);
    thrust::copy(l_sigma.begin(), l_sigma.end(), S);

    return true;
}

bool svdDecomposition_Vt_S(float* dataOverwrittenVt, float *S, int m, int n)
{
    const int l_count = m * n;
    const int l_minDim = std::min(m,n);

    culaStatus l_status;
    thrust::device_vector<float> l_dataOvt(dataOverwrittenVt, dataOverwrittenVt + l_count);
    thrust::device_vector<float> l_sigma(l_minDim);

    if ((l_status = culaDeviceSgesvd('O', 'N', m, n, l_dataOvt.data().get(), m, l_sigma.data().get(),NULL, m, NULL, n)) != culaNoError)
    {
        printf("%s\n", culaGetStatusString(l_status));
        return false;
    }

    thrust::copy(l_dataOvt.begin(), l_dataOvt.end(), dataOverwrittenVt);
    thrust::copy(l_sigma.begin(), l_sigma.end(), S);

    return true;
}

bool svdDecomposition_U_S(float* dataOverwrittedU, float *S, int m, int n)
{
    const int l_count = m * n;
    const int l_minDim = std::min(m,n);

    culaStatus l_status;
    thrust::device_vector<float> l_dataOU(dataOverwrittedU, dataOverwrittedU + l_count);
    thrust::device_vector<float> l_sigma(l_minDim);

    if ((l_status = culaDeviceSgesvd('N', 'O', m, n, l_dataOU.data().get(), m, l_sigma.data().get(),NULL, m, NULL, n)) != culaNoError)
    {
        printf("%s\n", culaGetStatusString(l_status));
        return false;
    }

    thrust::copy(l_dataOU.begin(), l_dataOU.end(), dataOverwrittedU);
    thrust::copy(l_sigma.begin(), l_sigma.end(), S);

    return true;
}

bool svdDecomposition_S(float* data, float *S, int m, int n)
{
    const int l_count = m * n;
    const int l_minDim = std::min(m,n);

    culaStatus l_status;
    thrust::device_vector<float> l_data(data, data + l_count);
    thrust::device_vector<float> l_sigma(l_minDim);

    if ((l_status = culaDeviceSgesvd('N', 'N', m, n, l_data.data().get(), m, l_sigma.data().get(),NULL, m, NULL, n)) != culaNoError)
    {
        printf("%s\n", culaGetStatusString(l_status));
        return false;
    }

    thrust::copy(l_sigma.begin(), l_sigma.end(), S);

    // JOBU != ‘O’ and JOBVT != ‘O’, the contents of A are destroyed.
    data = NULL;

    return true;
}


// ############################################################################################# LU INVERSION

//// Get a matrix element
//__device__ float GetElement(const Matrix &A, int row, int col)
//{
//    return A.elements[row * A.stride + col];
//}

//// Get a matrix element
//__device__ float GetElement(const MatrixD &A, int row, int col)
//{
//    return A.elements[row * A.stride + col];
//}

//// Set a matrix element
//__device__ void SetElement(Matrix &A, int row, int col,
//                           float value)
//{
//    A.elements[row * A.stride + col] = value;
//}

//// Set a matrix element
//__device__ void SetElement(MatrixD &A, int row, int col,
//                           float value)
//{
//    A.elements[row * A.stride + col] = value;
//}


//// Get the BLOCK_SIZExBLOCK_SIZE sub-matrix Asub of A that is
//// located col sub-matrices to the right and row sub-matrices down
//// from the upper-left corner of A
// __device__ Matrix GetSubMatrix(Matrix &A, int row, int col)
//{
//    Matrix Asub;
//    Asub.width    = BLOCKSIZE;
//    Asub.height   = BLOCKSIZE;
//    Asub.stride   = A.stride;
//    Asub.elements = &A.elements[A.stride * BLOCKSIZE * row
//                                         + BLOCKSIZE * col];
//    return Asub;
//}

// // Get the BLOCK_SIZExBLOCK_SIZE sub-matrix Asub of A that is
// // located col sub-matrices to the right and row sub-matrices down
// // from the upper-left corner of A
//  __device__ MatrixD GetSubMatrix(MatrixD &A, int row, int col)
// {
//     MatrixD Asub;
//     Asub.width    = BLOCKSIZE;
//     Asub.height   = BLOCKSIZE;
//     Asub.stride   = A.stride;
//     Asub.elements = &A.elements[A.stride * BLOCKSIZE * row
//                                          + BLOCKSIZE * col];
//     return Asub;
// }

//// Matrix multiplication kernel called by MatMul()
// __global__ void MatMulKernel(Matrix &A, Matrix &B, Matrix &C)
//{
//    // Block row and column
//    int blockRow = blockIdx.y;
//    int blockCol = blockIdx.x;

//    // Each thread block computes one sub-matrix Csub of C
//    Matrix Csub = GetSubMatrix(C, blockRow, blockCol);

//    // Each thread computes one element of Csub
//    // by accumulating results into Cvalue
//    float Cvalue = 0;

//    // Thread row and column within Csub
//    int row = threadIdx.y;
//    int col = threadIdx.x;

//    // Loop over all the sub-matrices of A and B that are
//    // required to compute Csub
//    // Multiply each pair of sub-matrices together
//    // and accumulate the results
//    for (int m = 0; m < (A.width / BLOCKSIZE); ++m) {

//        // Get sub-matrix Asub of A
//        Matrix Asub = GetSubMatrix(A, blockRow, m);

//        // Get sub-matrix Bsub of B
//        Matrix Bsub = GetSubMatrix(B, m, blockCol);

//        // Shared memory used to store Asub and Bsub respectively
//        __shared__ float As[BLOCKSIZE][BLOCKSIZE];
//        __shared__ float Bs[BLOCKSIZE][BLOCKSIZE];

//        // Load Asub and Bsub from device memory to shared memory
//        // Each thread loads one element of each sub-matrix
//        As[row][col] = GetElement(Asub, row, col);
//        Bs[row][col] = GetElement(Bsub, row, col);

//        // Synchronize to make sure the sub-matrices are loaded
//        // before starting the computation
//        __syncthreads();

//        // Multiply Asub and Bsub together
//        for (int e = 0; e < BLOCKSIZE; ++e)
//            Cvalue += As[row][e] * Bs[e][col];

//        // Synchronize to make sure that the preceding
//        // computation is done before loading two new
//        // sub-matrices of A and B in the next iteration
//        __syncthreads();
//    }

//    // Write Csub to device memory
//    // Each thread writes one element
//    SetElement(Csub, row, col, Cvalue);
//}

// // Matrix multiplication kernel called by MatMul()
//  __global__ void MatMulKernel(MatrixD &A, MatrixD &B, MatrixD &C)
// {
//     // Block row and column
//     int blockRow = blockIdx.y;
//     int blockCol = blockIdx.x;

//     // Each thread block computes one sub-matrix Csub of C
//     MatrixD Csub = GetSubMatrix(C, blockRow, blockCol);

//     // Each thread computes one element of Csub
//     // by accumulating results into Cvalue
//     float Cvalue = 0;

//     // Thread row and column within Csub
//     int row = threadIdx.y;
//     int col = threadIdx.x;

//     // Loop over all the sub-matrices of A and B that are
//     // required to compute Csub
//     // Multiply each pair of sub-matrices together
//     // and accumulate the results
//     for (int m = 0; m < (A.width / BLOCKSIZE); ++m) {

//         // Get sub-matrix Asub of A
//         MatrixD Asub = GetSubMatrix(A, blockRow, m);

//         // Get sub-matrix Bsub of B
//         MatrixD Bsub = GetSubMatrix(B, m, blockCol);

//         // Shared memory used to store Asub and Bsub respectively
//         __shared__ double As[BLOCKSIZE][BLOCKSIZE];
//         __shared__ double Bs[BLOCKSIZE][BLOCKSIZE];

//         // Load Asub and Bsub from device memory to shared memory
//         // Each thread loads one element of each sub-matrix
//         As[row][col] = GetElement(Asub, row, col);
//         Bs[row][col] = GetElement(Bsub, row, col);

//         // Synchronize to make sure the sub-matrices are loaded
//         // before starting the computation
//         __syncthreads();

//         // Multiply Asub and Bsub together
//         for (int e = 0; e < BLOCKSIZE; ++e)
//             Cvalue += As[row][e] * Bs[e][col];

//         // Synchronize to make sure that the preceding
//         // computation is done before loading two new
//         // sub-matrices of A and B in the next iteration
//         __syncthreads();
//     }

//     // Write Csub to device memory
//     // Each thread writes one element
//     SetElement(Csub, row, col, Cvalue);
// }

// // Matrix multiplication - Host code
// // Matrix dimensions are assumed to be multiples of BLOCK_SIZE
// void matMult(const Matrix &A, const Matrix &B, Matrix &C, const int blockSize)
// {
//     // Load A and B to device memory
//     Matrix d_A;
//     d_A.width = d_A.stride = A.width; d_A.height = A.height;
//     size_t size = A.width * A.height * sizeof(float);
//     hipMalloc(&d_A.elements, size);
//     hipMemcpy(d_A.elements, A.elements, size,
//                hipMemcpyHostToDevice);
//     Matrix d_B;
//     d_B.width = d_B.stride = B.width; d_B.height = B.height;
//     size = B.width * B.height * sizeof(float);
//     hipMalloc(&d_B.elements, size);
//     hipMemcpy(d_B.elements, B.elements, size,
//     hipMemcpyHostToDevice);

//     // Allocate C in device memory
//     Matrix d_C;
//     d_C.width = d_C.stride = C.width; d_C.height = C.height;
//     size = C.width * C.height * sizeof(float);
//     hipMalloc(&d_C.elements, size);

//     // Invoke kernel
//     dim3 dimBlock(blockSize, blockSize);
//     dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
//     MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

//     // Read C from device memory
//     hipMemcpy(C.elements, d_C.elements, size,
//                hipMemcpyDeviceToHost);

//     // Free device memory
//     hipFree(d_A.elements);
//     hipFree(d_B.elements);
//     hipFree(d_C.elements);
// }

// // Matrix multiplication - Host code
// // Matrix dimensions are assumed to be multiples of BLOCK_SIZE
// void matMultD(const MatrixD &A, const MatrixD &B, MatrixD &C, const int blockSize)
// {
//     // Load A and B to device memory
//     MatrixD d_A;
//     d_A.width = d_A.stride = A.width; d_A.height = A.height;
//     size_t size = A.width * A.height * sizeof(double);
//     hipMalloc(&d_A.elements, size);
//     hipMemcpy(d_A.elements, A.elements, size,
//                hipMemcpyHostToDevice);
//     MatrixD d_B;
//     d_B.width = d_B.stride = B.width; d_B.height = B.height;
//     size = B.width * B.height * sizeof(double);
//     hipMalloc(&d_B.elements, size);
//     hipMemcpy(d_B.elements, B.elements, size,
//     hipMemcpyHostToDevice);

//     // Allocate C in device memory
//     MatrixD d_C;
//     d_C.width = d_C.stride = C.width; d_C.height = C.height;
//     size = C.width * C.height * sizeof(double);
//     hipMalloc(&d_C.elements, size);

//     // Invoke kernel
//     dim3 dimBlock(blockSize, blockSize);
//     dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
//     MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

//     // Read C from device memory
//     hipMemcpy(C.elements, d_C.elements, size,
//                hipMemcpyDeviceToHost);

//     // Free device memory
//     hipFree(d_A.elements);
//     hipFree(d_B.elements);
//     hipFree(d_C.elements);
// }
