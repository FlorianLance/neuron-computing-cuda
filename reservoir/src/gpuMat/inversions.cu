#include "hip/hip_runtime.h"
/*******************************************************************************
**                                                                            **
**  Language Learning - Reservoir Computing - GPU                             **
**  An interface for language learning with neuron computing using GPU        **
**  acceleration.                                                             **
**                                                                            **
**  This program is free software: you can redistribute it and/or modify      **
**  it under the terms of the GNU Lesser General Public License as published  **
**  by the Free Software Foundation, either version 3 of the License, or      **
**  (at your option) any later version.                                       **
**                                                                            **
**  This program is distributed in the hope that it will be useful,           **
**  but WITHOUT ANY WARRANTY; without even the implied warranty of            **
**  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the             **
**  GNU Lesser General Public License for more details.                       **
**                                                                            **
**  You should have received a copy of the GNU Lesser General Public License  **
**  along with Foobar.  If not, see <http://www.gnu.org/licenses/>.           **
**                                                                            **
********************************************************************************/


/**
 * \file inversions.cu
 * \brief defines cuda matrix inversion functions/
 * \author Florian Lance
 * \date 01/10/14
 */

#include "cula.h"

#include "gpuMat/configCuda.h"


#include <stdio.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>


int culaWarmup(int gpuDevice)
{
    int cudaMinimumVersion   = culaGetCudaMinimumVersion();
    int cudaRuntimeVersion   = culaGetCudaRuntimeVersion();
    int cudaDriverVersion    = culaGetCudaDriverVersion();
    int cublasMinimumVersion = culaGetCublasMinimumVersion();
    int cublasRuntimeVersion = culaGetCublasRuntimeVersion();

    if(cudaRuntimeVersion < cudaMinimumVersion)
    {
        printf("-1\n %d", cudaMinimumVersion);
        return -1;
    }

    if(cudaDriverVersion < cudaMinimumVersion)
    {
        printf("-2\n %d", cudaMinimumVersion);
        return -1;
    }

    if(cublasRuntimeVersion < cublasMinimumVersion)
    {
        printf("-3\n %d", cublasMinimumVersion);
        return -1;
    }

    culaStatus l_oStatus;
    char l_buf[256];
    int l_info;

    culaSelectDevice(gpuDevice);
    l_oStatus = culaInitialize();

    if(l_oStatus != culaNoError)
    {
        l_info = culaGetErrorInfo();

        culaGetErrorInfoString(l_oStatus, l_info, l_buf, sizeof(l_buf));

        printf("culaWarmup : %s\n", culaGetStatusString(l_oStatus));
        printf("%s\n", l_buf);
        return -1;
    }

    return 0;
}

void culaStop()
{
    culaShutdown();
}

// ############################################################################################# SVD DECOMPOSITION

bool svdDecomposition(float* data, int m, int n, float* S, float* VT, float* U)
{
    const int l_count = m * n;
    const int l_minDim = std::min(m,n);
    const char l_jobu  = 'A';
    const char l_jobvt = 'A';

    culaStatus l_status;

    thrust::device_vector<float> l_data(data, data + l_count);
    thrust::device_vector<float> l_U(m * m);
    thrust::device_vector<float> l_sigma(l_minDim);
    thrust::device_vector<float> l_Vt(n * n);

    if ((l_status = culaDeviceSgesvd(l_jobu, l_jobvt, m, n, l_data.data().get(), m, l_sigma.data().get(),l_U.data().get(), m, l_Vt.data().get(), n)) != culaNoError)
    {
        printf("svdDecomposition : %s\n", culaGetStatusString(l_status));
        return false;
    }

    delete[] data;
    data = NULL;

    thrust::copy(l_U.begin(), l_U.end(), U);
    thrust::copy(l_Vt.begin(), l_Vt.end(), VT);
    thrust::copy(l_sigma.begin(), l_sigma.end(), S);

    return true;
}

bool svdDecomposition_all(float* dataOverwrittenVt, int m, int n, float* S, float* U)
{
    const int l_count = m * n;
    const int l_minDim = std::min(m,n);

    culaStatus l_status;
    thrust::device_vector<float> l_dataOvt(dataOverwrittenVt, dataOverwrittenVt + l_count);
    thrust::device_vector<float> l_U(m * m);
    thrust::device_vector<float> l_sigma(l_minDim);

    if ((l_status = culaDeviceSgesvd('A', 'O', m, n, l_dataOvt.data().get(), m, l_sigma.data().get(),l_U.data().get(), m, NULL, n)) != culaNoError)
    {
        printf("svdDecomposition_all : %s\n", culaGetStatusString(l_status));
        return false;
    }

    thrust::copy(l_dataOvt.begin(), l_dataOvt.end(), dataOverwrittenVt);
    thrust::copy(l_U.begin(), l_U.end(), U);
    thrust::copy(l_sigma.begin(), l_sigma.end(), S);

    return true;
}

bool svdDecomposition_Vt_S(float* dataOverwrittenVt, float *S, int m, int n)
{
    const int l_count = m * n;
    const int l_minDim = std::min(m,n);

    culaStatus l_status;
    thrust::device_vector<float> l_dataOvt(dataOverwrittenVt, dataOverwrittenVt + l_count);
    thrust::device_vector<float> l_sigma(l_minDim);

    if ((l_status = culaDeviceSgesvd('O', 'N', m, n, l_dataOvt.data().get(), m, l_sigma.data().get(),NULL, m, NULL, n)) != culaNoError)
    {
        printf("svdDecomposition_Vt_S : %s\n", culaGetStatusString(l_status));
        return false;
    }

    thrust::copy(l_dataOvt.begin(), l_dataOvt.end(), dataOverwrittenVt);
    thrust::copy(l_sigma.begin(), l_sigma.end(), S);

    return true;
}

bool svdDecomposition_U_S(float* dataOverwrittedU, float *S, int m, int n)
{
    const int l_count = m * n;
    const int l_minDim = std::min(m,n);

    culaStatus l_status;
    thrust::device_vector<float> l_dataOU(dataOverwrittedU, dataOverwrittedU + l_count);
    thrust::device_vector<float> l_sigma(l_minDim);

    if ((l_status = culaDeviceSgesvd('N', 'O', m, n, l_dataOU.data().get(), m, l_sigma.data().get(),NULL, m, NULL, n)) != culaNoError)
    {
        printf("svdDecomposition_U_S : %s\n", culaGetStatusString(l_status));
        return false;
    }

    thrust::copy(l_dataOU.begin(), l_dataOU.end(), dataOverwrittedU);
    thrust::copy(l_sigma.begin(), l_sigma.end(), S);

    return true;
}

bool svdDecomposition_S(float* data, float *S, int m, int n)
{
    const int l_count = m * n;
    const int l_minDim = std::min(m,n);

    culaStatus l_status;
    thrust::device_vector<float> l_data(data, data + l_count);
    thrust::device_vector<float> l_sigma(l_minDim);

    if ((l_status = culaDeviceSgesvd('N', 'N', m, n, l_data.data().get(), m, l_sigma.data().get(),NULL, m, NULL, n)) != culaNoError)
    {
        printf("svdDecomposition_S : %s\n", culaGetStatusString(l_status));
        return false;
    }

    thrust::copy(l_sigma.begin(), l_sigma.end(), S);

    // JOBU != ‘O’ and JOBVT != ‘O’, the contents of A are destroyed.
    data = NULL;

    return true;
}
