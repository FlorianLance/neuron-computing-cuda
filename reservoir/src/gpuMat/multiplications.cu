#include "hip/hip_runtime.h"

/*******************************************************************************
**                                                                            **
**  Language Learning - Reservoir Computing - GPU                             **
**  An interface for language learning with neuron computing using GPU        **
**  acceleration.                                                             **
**                                                                            **
**  This program is free software: you can redistribute it and/or modify      **
**  it under the terms of the GNU Lesser General Public License as published  **
**  by the Free Software Foundation, either version 3 of the License, or      **
**  (at your option) any later version.                                       **
**                                                                            **
**  This program is distributed in the hope that it will be useful,           **
**  but WITHOUT ANY WARRANTY; without even the implied warranty of            **
**  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the             **
**  GNU Lesser General Public License for more details.                       **
**                                                                            **
**  You should have received a copy of the GNU Lesser General Public License  **
**  along with Foobar.  If not, see <http://www.gnu.org/licenses/>.           **
**                                                                            **
********************************************************************************/

/**
 * \file multiplications.cu
 * \brief defines cuda matrix multiplications functions/
 * \author Florian Lance
 * \date 01/10/14
 */

#include "gpuMat/configCuda.h"

#include <stdio.h>
#include <hipblas.h>

// slower than opencv, useless
void vectorSquareMatrixMult(const MatrixD &matA, const MatrixD &vecB, MatrixD &res)
{
    // device pointers
    double *l_dataMatA, *l_dataVecB, *l_dataTemp;

    int l_N = matA.height;
    hipMalloc((void**)&l_dataMatA, l_N * l_N * sizeof(double));
    hipMalloc((void**)&l_dataTemp, l_N *     sizeof(double));
    hipMalloc((void**)&l_dataVecB, l_N *     sizeof(double));

    hipblasSetVector(l_N,    sizeof(double), vecB.elements, 1, l_dataVecB, 1);
    hipblasSetMatrix(l_N, l_N, sizeof(double), matA.elements, l_N, l_dataMatA, l_N);

    hipblasHandle_t l_handle;
    hipblasCreate(&l_handle);

    double l_alpha = 1.0f;
    double l_beta  = 0.0f;
    hipblasDgemv(l_handle, HIPBLAS_OP_T, l_N, l_N, &l_alpha, l_dataMatA, l_N, l_dataVecB, 1, &l_beta, l_dataTemp, 1);
    hipblasGetVector(l_N, sizeof(double), l_dataTemp, 1, res.elements, 1);

    hipFree(l_dataMatA);
    hipFree(l_dataVecB);
    hipFree(l_dataTemp);
}




//////////////////////////////////////////////////////
//! Matrix multiplication on the device: C = A * B
//! wA is A's width and wB is B's width
//////////////////////////////////////////////////////
__global__ void
matrixMul( float* C, float* A, float* B, int wA, int wB)
{
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Index of the first sub-matrix of A processed
    // by the block
    int aBegin = wA * BLOCKSIZE * by;

    // Index of the last sub-matrix of A processed
    // by the block
    int aEnd   = aBegin + wA - 1;

    // Step size used to iterate through the
    // sub-matrices of A
    int aStep  = BLOCKSIZE;

    // Index of the first sub-matrix of B processed
    // by the block
    int bBegin = BLOCKSIZE * bx;

    // Step size used to iterate through the
    // sub-matrices of B
    int bStep  = BLOCKSIZE * wB;

    float Csub = 0.f;

    // Loop over all the sub-matrices of A and B
    // required to compute the block sub-matrix
    for (int a = aBegin, b = bBegin;
             a <= aEnd;
             a += aStep, b += bStep)
    {

        // Declaration of the shared memory array As
        // used to store the sub-matrix of A
        __shared__ float As[BLOCKSIZE][BLOCKSIZE];

        // Declaration of the shared memory array Bs
        // used to store the sub-matrix of B
        __shared__ float Bs[BLOCKSIZE][BLOCKSIZE];

        // Load the matrices from global memory
        // to shared memory; each thread loads
        // one element of each matrix
        As[ty][tx] = A[a + wA * ty + tx];
        Bs[ty][tx] = B[b + wB * ty + tx];

        // Synchronize to make sure the matrices
        // are loaded
        __syncthreads();

        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
        for (int k = 0; k < BLOCKSIZE; ++k)
            Csub += As[ty][k] * Bs[k][tx];

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();

    }

    // Write the block sub-matrix to device memory;
    // each thread writes one element
    int c = wB * BLOCKSIZE * by + BLOCKSIZE * bx;
    C[c + wB * ty + tx] = Csub;

}


 void matMult3(const Matrix &A, const Matrix &B, Matrix &C)
 {

   // 1. allocate host memory for matrices A and B
   unsigned int size_A = A.width * A.height;
   unsigned int mem_size_A = sizeof(float) * size_A;
//    float* h_A = (float*) malloc(mem_size_A);

   unsigned int size_B = B.width * B.height;
   unsigned int mem_size_B = sizeof(float) * size_B;
//    float* h_B = (float*) malloc(mem_size_B);

   // 2. initialize host memory
//    randomInit(h_A, size_A);
//    randomInit(h_B, size_B);

   // 8. allocate device memory
   float* d_A;
   float* d_B;
   hipMalloc((void**) &d_A, mem_size_A);
   hipMalloc((void**) &d_B, mem_size_B);

   // 9. copy host memory to device
   hipMemcpy(d_A, A.elements, mem_size_A,
   hipMemcpyHostToDevice);
   hipMemcpy(d_B, B.elements, mem_size_B,
   hipMemcpyHostToDevice);

   // 4. allocate host memory for the result C
   unsigned int size_C = C.width * C.height;
   unsigned int mem_size_C = sizeof(float) * size_C;
//    float* h_C = (float*) malloc(mem_size_C);

   // 10. allocate device memory for the result
   float* d_C;
   hipMalloc((void**) &d_C, mem_size_C);

   // 5. perform the calculation
   // setup execution parameters
   dim3 threads(BLOCKSIZE, BLOCKSIZE);
   dim3 grid(C.width / threads.x, C.height / threads.y);

   // execute the kernel
   matrixMul<<< grid, threads >>>(d_C, d_A,
                                  d_B, A.width, B.width);

   hipFree(d_A);
   hipFree(d_B);

   // 11. copy result from device to host
   hipMemcpy(C.elements, d_C, mem_size_C,
   hipMemcpyDeviceToHost);

   // 7. clean up memory
   hipFree(d_C);
}







 //////////////////////////////////////////////////////
 //! Matrix multiplication on the device: C = A * B
 //! wA is A's width and wB is B's width
 //////////////////////////////////////////////////////
 __global__ void
 matrixMul( double* C, double* A, double* B, int wA, int wB)
 {
     // Block index
     int bx = blockIdx.x;
     int by = blockIdx.y;

     // Thread index
     int tx = threadIdx.x;
     int ty = threadIdx.y;

     // Index of the first sub-matrix of A processed
     // by the block
     int aBegin = wA * BLOCKSIZE * by;

     // Index of the last sub-matrix of A processed
     // by the block
     int aEnd   = aBegin + wA - 1;

     // Step size used to iterate through the
     // sub-matrices of A
     int aStep  = BLOCKSIZE;

     // Index of the first sub-matrix of B processed
     // by the block
     int bBegin = BLOCKSIZE * bx;

     // Step size used to iterate through the
     // sub-matrices of B
     int bStep  = BLOCKSIZE * wB;

     double Csub = 0.f;

     // Loop over all the sub-matrices of A and B
     // required to compute the block sub-matrix
     for (int a = aBegin, b = bBegin;
              a <= aEnd;
              a += aStep, b += bStep)
     {

         // Declaration of the shared memory array As
         // used to store the sub-matrix of A
         __shared__ double As[BLOCKSIZE][BLOCKSIZE];

         // Declaration of the shared memory array Bs
         // used to store the sub-matrix of B
         __shared__ double Bs[BLOCKSIZE][BLOCKSIZE];

         // Load the matrices from global memory
         // to shared memory; each thread loads
         // one element of each matrix
         As[ty][tx] = A[a + wA * ty + tx];
         Bs[ty][tx] = B[b + wB * ty + tx];

         // Synchronize to make sure the matrices
         // are loaded
         __syncthreads();

         // Multiply the two matrices together;
         // each thread computes one element
         // of the block sub-matrix
         for (int k = 0; k < BLOCKSIZE; ++k)
             Csub += As[ty][k] * Bs[k][tx];

         // Synchronize to make sure that the preceding
         // computation is done before loading two new
         // sub-matrices of A and B in the next iteration
         __syncthreads();

     }

     // Write the block sub-matrix to device memory;
     // each thread writes one element
     int c = wB * BLOCKSIZE * by + BLOCKSIZE * bx;
     C[c + wB * ty + tx] = Csub;

 }


 void matMult3D(const MatrixD &A, const MatrixD &B, MatrixD &C)
 {

   // 1. allocate host memory for matrices A and B
   unsigned int size_A = A.width * A.height;
   unsigned int mem_size_A = sizeof(double) * size_A;

   unsigned int size_B = B.width * B.height;
   unsigned int mem_size_B = sizeof(double) * size_B;

   // 8. allocate device memory
   double* d_A;
   double* d_B;
   hipMalloc((void**) &d_A, mem_size_A);
   hipMalloc((void**) &d_B, mem_size_B);

   // 9. copy host memory to device
   hipMemcpy(d_A, A.elements, mem_size_A,
   hipMemcpyHostToDevice);
   hipMemcpy(d_B, B.elements, mem_size_B,
   hipMemcpyHostToDevice);

   // 4. allocate host memory for the result C
   unsigned int size_C = C.width * C.height;
   unsigned int mem_size_C = sizeof(double) * size_C;

   // 10. allocate device memory for the result
   double* d_C;
   hipMalloc((void**) &d_C, mem_size_C);

   // 5. perform the calculation
   // setup execution parameters
   dim3 threads(BLOCKSIZE, BLOCKSIZE);
   dim3 grid(C.width / threads.x, C.height / threads.y);

   // execute the kernel
   matrixMul<<< grid, threads >>>(d_C, d_A,
                                  d_B, A.width, B.width);

   hipFree(d_A);
   hipFree(d_B);

   // 11. copy result from device to host
   hipMemcpy(C.elements, d_C, mem_size_C,
   hipMemcpyDeviceToHost);

   // 7. clean up memory
   hipFree(d_C);
}


